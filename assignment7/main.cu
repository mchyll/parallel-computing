#include "hip/hip_runtime.h"
/*
PARALLEL COMPUTING - ASSIGNMENT 7
Magnus Conrad Hyll
*/

#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <sys/time.h>
extern "C" {
    #include "libs/bitmap.h"
}
#include <hip/hip_runtime_api.h>

#define ERROR_EXIT -1

#define BLOCK_WIDTH 8
#define BLOCK_HEIGHT 8

#define cudaErrorCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %s %d\n", hipGetErrorName(code), hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

// Convolutional Filter Examples, each with dimension 3,
// gaussian filter with dimension 5
// If you apply another filter, remember not only to exchange
// the filter but also the filterFactor and the correct dimension.

int const sobelYFilter[] = {-1, -2, -1,
                             0,  0,  0,
                             1,  2,  1};
float const sobelYFilterFactor = (float) 1.0;

int const sobelXFilter[] = {1, -0, -1,
                            2,  0, -2,
                            1,  0, -1 , 0};
float const sobelXFilterFactor = (float) 1.0;


int const laplacian1Filter[] = {  -1,  -4,  -1,
                                 -4,  20,  -4,
                                 -1,  -4,  -1};

float const laplacian1FilterFactor = (float) 1.0;

int const laplacian2Filter[] = { 0,  1,  0,
                                 1, -4,  1,
                                 0,  1,  0};
float const laplacian2FilterFactor = (float) 1.0;

int const laplacian3Filter[] = { -1,  -1,  -1,
                                  -1,   8,  -1,
                                  -1,  -1,  -1};
float const laplacian3FilterFactor = (float) 1.0;


//Bonus Filter:

int const gaussianFilter[] = { 1,  4,  6,  4, 1,
                               4, 16, 24, 16, 4,
                               6, 24, 36, 24, 6,
                               4, 16, 24, 16, 4,
                               1,  4,  6,  4, 1 };

float const gaussianFilterFactor = (float) 1.0 / 256.0;


// Apply convolutional filter on image data
void applyFilter(unsigned char **out, unsigned char **in, unsigned int width, unsigned int height, const int *filter, unsigned int filterDim, float filterFactor) {
  unsigned int const filterCenter = (filterDim / 2);
  for (unsigned int y = 0; y < height; y++) {
    for (unsigned int x = 0; x < width; x++) {
      int aggregate = 0;
      for (unsigned int ky = 0; ky < filterDim; ky++) {
        int nky = filterDim - 1 - ky;
        for (unsigned int kx = 0; kx < filterDim; kx++) {
          int nkx = filterDim - 1 - kx;

          int yy = y + (ky - filterCenter);
          int xx = x + (kx - filterCenter);
          if (xx >= 0 && xx < (int) width && yy >=0 && yy < (int) height)
            aggregate += in[yy][xx] * filter[nky * filterDim + nkx];
        }
      }
      aggregate *= filterFactor;
      if (aggregate > 0) {
        out[y][x] = (aggregate > 255) ? 255 : aggregate;
      } else {
        out[y][x] = 0;
      }
    }
  }
}

// Apply convolutional filter on image data
__global__ void cudaApplyFilter(unsigned char *out, unsigned char *in, unsigned int width, unsigned int height, int *filter, unsigned int filterDim, float filterFactor) {
  unsigned int const filterCenter = (filterDim / 2);
  int x = threadIdx.x + blockIdx.x * blockDim.x;  // x coordinate of pixel
  int y = threadIdx.y + blockIdx.y * blockDim.y;  // y coordinate of pixel

  // Check if point is within image
  if (x < width && y < height) {
    int aggregate = 0;
    for (unsigned int ky = 0; ky < filterDim; ky++) {
      int nky = filterDim - 1 - ky;
      for (unsigned int kx = 0; kx < filterDim; kx++) {
        int nkx = filterDim - 1 - kx;

        int yy = y + (ky - filterCenter);
        int xx = x + (kx - filterCenter);
        if (xx >= 0 && xx < (int) width && yy >=0 && yy < (int) height)
          aggregate += in[yy * width + xx] * filter[nky * filterDim + nkx];
      }
    }
    aggregate *= filterFactor;
    if (aggregate > 0) {
      out[y * width + x] = (aggregate > 255) ? 255 : aggregate;
    } else {
      out[y * width + x] = 0;
    }
  }
}

// Apply convolutional filter on image data
__global__ void cudaApplyFilterSharedMem(unsigned char *out, unsigned char *in, unsigned int width, unsigned int height, int *filter, unsigned int filterDim, float filterFactor) {
  unsigned int const filterCenter = (filterDim / 2);
  int x = threadIdx.x + blockIdx.x * blockDim.x;  // x coordinate of pixel
  int y = threadIdx.y + blockIdx.y * blockDim.y;  // y coordinate of pixel

  // The thread block cache containing part of the image
  __shared__ unsigned char blockCache[BLOCK_WIDTH * BLOCK_HEIGHT];
  // The filter, cached to shared memory. The size of this array is determined at kernel launch
  extern __shared__ int filterCache[];

  // Copy filter to shared memory
  if (threadIdx.x < filterDim && threadIdx.y < filterDim)
    filterCache[threadIdx.y * filterDim + threadIdx.x] = filter[threadIdx.y * filterDim + threadIdx.x];

  // Copy image block to shared memory
  if (x < width && y < height)
    blockCache[threadIdx.y * blockDim.x + threadIdx.x] = in[y * width + x];

  // Wait until all threads have finished copying to shared mem
  __syncthreads();

  // Check if point is within image
  if (x < width && y < height) {
    int aggregate = 0;
    for (unsigned int ky = 0; ky < filterDim; ky++) {
      int nky = filterDim - 1 - ky;
      for (unsigned int kx = 0; kx < filterDim; kx++) {
        int nkx = filterDim - 1 - kx;

        // Check first if current pixel is within block cache
        int block_x = threadIdx.x + (kx - filterCenter);
        int block_y = threadIdx.y + (ky - filterCenter);
        if (block_x >= 0 && block_x < BLOCK_WIDTH && block_y >= 0 && block_y < BLOCK_HEIGHT) {
          aggregate += blockCache[block_y * BLOCK_WIDTH + block_x] * filterCache[nky * filterDim + nkx];
        }
        // If not, do the usual and read from global memory
        else {
          int yy = y + (ky - filterCenter);
          int xx = x + (kx - filterCenter);
          if (xx >= 0 && xx < (int) width && yy >=0 && yy < (int) height)
            aggregate += in[yy * width + xx] * filterCache[nky * filterDim + nkx];
        }
      }
    }
    aggregate *= filterFactor;
    if (aggregate > 0) {
      out[y * width + x] = (aggregate > 255) ? 255 : aggregate;
    } else {
      out[y * width + x] = 0;
    }
  } 
}

void help(char const *exec, char const opt, char const *optarg) {
    FILE *out = stdout;
    if (opt != 0) {
        out = stderr;
        if (optarg) {
            fprintf(out, "Invalid parameter - %c %s\n", opt, optarg);
        } else {
            fprintf(out, "Invalid parameter - %c\n", opt);
        }
    }
    fprintf(out, "%s [options] <input-bmp> <output-bmp>\n", exec);
    fprintf(out, "\n");
    fprintf(out, "Options:\n");
    fprintf(out, "  -i, --iterations <iterations>    number of iterations (1)\n");

    fprintf(out, "\n");
    fprintf(out, "Example: %s in.bmp out.bmp -i 10000\n", exec);
}

void runSerial(bmpImageChannel* imageChannel, int iterations, const int* filter, int filterDim, float filterFactor) {
  //Here we do the actual computation!
  // imageChannel->data is a 2-dimensional array of unsigned char which is accessed row first ([y][x])
  bmpImageChannel *processImageChannel = newBmpImageChannel(imageChannel->width, imageChannel->height);
  for (unsigned int i = 0; i < iterations; i ++) {
    applyFilter(processImageChannel->data,
                imageChannel->data,
                imageChannel->width,
                imageChannel->height,
                filter, filterDim, filterFactor);

    // Swap the data pointers
    unsigned char ** tmp = processImageChannel->data;
    processImageChannel->data = imageChannel->data;
    imageChannel->data = tmp;
    unsigned char * tmp_raw = processImageChannel->rawdata;
    processImageChannel->rawdata = imageChannel->rawdata;
    imageChannel->rawdata = tmp_raw;
  }
  freeBmpImageChannel(processImageChannel);
}

void runCuda(bmpImageChannel* imageChannel, int iterations, const int* filter, int filterDim, float filterFactor) {
  // Allocate memory for input image, output image and filter on GPU
  unsigned char* inImage;
  cudaErrorCheck(hipMalloc((void**) &inImage, imageChannel->width * imageChannel->height * sizeof(int)));
  unsigned char* outImage;
  cudaErrorCheck(hipMalloc((void**) &outImage, imageChannel->width * imageChannel->height * sizeof(int)));
  int* deviceFilter;
  cudaErrorCheck(hipMalloc((void**) &deviceFilter, filterDim * filterDim * sizeof(int)));

  // Copy data for original image and filter to GPU
  cudaErrorCheck(hipMemcpy(inImage, imageChannel->rawdata, imageChannel->width * imageChannel->height * sizeof(int), hipMemcpyHostToDevice));
  cudaErrorCheck(hipMemcpy(deviceFilter, filter, filterDim * filterDim * sizeof(int), hipMemcpyHostToDevice));

  // Define dimensions for block-grid and thread-blocks
  dim3 gridDim(imageChannel->width / BLOCK_WIDTH + 1, imageChannel->height / BLOCK_HEIGHT + 1); // Grid consists of blocks
  dim3 blockDim(BLOCK_WIDTH, BLOCK_HEIGHT); // Block consists of threads

  // Here we do the actual computation!
  for (unsigned int i = 0; i < iterations; i++) {
    // Thrid launch argument is the size of the array in shared memory containing the filter
    cudaApplyFilterSharedMem<<<gridDim, blockDim, (filterDim*filterDim)>>>(outImage, inImage, imageChannel->width, imageChannel->height, deviceFilter, filterDim, filterFactor);
    // cudaApplyFilter<<<gridDim, blockDim>>>(outImage, inImage, imageChannel->width, imageChannel->height, deviceFilter, filterDim, filterFactor);

    // Swap the data pointers
    unsigned char* tmp = inImage;
    inImage = outImage;
    outImage = tmp;
  }

  // Copy resulting image back to main memory
  cudaErrorCheck(hipMemcpy(imageChannel->rawdata, inImage, imageChannel->width * imageChannel->height * sizeof(int), hipMemcpyDeviceToHost));

  // Free the GPU-allocated memory
  cudaErrorCheck(hipFree(inImage));
  cudaErrorCheck(hipFree(outImage));
  cudaErrorCheck(hipFree(deviceFilter));
}

double walltime() {
	static struct timeval t;
	gettimeofday(&t, NULL);
	return t.tv_sec + 1e-6 * t.tv_usec;
}

int main(int argc, char **argv) {
  /*
    Parameter parsing, don't change this!
   */
  unsigned int iterations = 1;
  char *output = NULL;
  char *input = NULL;
  int ret = 0;

  static struct option const long_options[] =  {
      {"help",       no_argument,       0, 'h'},
      {"iterations", required_argument, 0, 'i'},
      {0, 0, 0, 0}
  };

  static char const * short_options = "hi:";
  {
    char *endptr;
    int c;
    int option_index = 0;
    while ((c = getopt_long(argc, argv, short_options, long_options, &option_index)) != -1) {
      switch (c) {
      case 'h':
        help(argv[0],0, NULL);
        return 0;
      case 'i':
        iterations = strtol(optarg, &endptr, 10);
        if (endptr == optarg) {
          help(argv[0], c, optarg);
          return ERROR_EXIT;
        }
        break;
      default:
        abort();
      }
    }
  }

  if (argc <= (optind+1)) {
    help(argv[0],' ',"Not enough arugments");
    return ERROR_EXIT;
  }
  input = (char *)calloc(strlen(argv[optind]) + 1, sizeof(char));
  strncpy(input, argv[optind], strlen(argv[optind]));
  optind++;

  output = (char *)calloc(strlen(argv[optind]) + 1, sizeof(char));
  strncpy(output, argv[optind], strlen(argv[optind]));
  optind++;

  /*
    End of Parameter parsing!
   */

  /*
    Create the BMP image and load it from disk.
   */
  bmpImage *image = newBmpImage(0,0);
  if (image == NULL) {
    fprintf(stderr, "Could not allocate new image!\n");
  }

  if (loadBmpImage(image, input) != 0) {
    fprintf(stderr, "Could not load bmp image '%s'!\n", input);
    freeBmpImage(image);
    return ERROR_EXIT;
  }


  // Create a single color channel image. It is easier to work just with one color
  bmpImageChannel *imageChannel = newBmpImageChannel(image->width, image->height);
  if (imageChannel == NULL) {
    fprintf(stderr, "Could not allocate new image channel!\n");
    freeBmpImage(image);
    return ERROR_EXIT;
  }

  // Extract from the loaded image an average over all colors - nothing else than
  // a black and white representation
  // extractImageChannel and mapImageChannel need the images to be in the exact
  // same dimensions!
  // Other prepared extraction functions are extractRed, extractGreen, extractBlue
  if(extractImageChannel(imageChannel, image, extractAverage) != 0) {
    fprintf(stderr, "Could not extract image channel!\n");
    freeBmpImage(image);
    freeBmpImageChannel(imageChannel);
    return ERROR_EXIT;
  }

  const int* filter = laplacian1Filter;
  int filterDim = 3;
  float filterFactor = laplacian1FilterFactor;

  double tStart = walltime();

  // runSerial(imageChannel, iterations, filter, filterDim, filterFactor);
  runCuda(imageChannel, iterations, filter, filterDim, filterFactor);

  printf("Time: %.5f sec\n", walltime() - tStart);

  // Map our single color image back to a normal BMP image with 3 color channels
  // mapEqual puts the color value on all three channels the same way
  // other mapping functions are mapRed, mapGreen, mapBlue
  if (mapImageChannel(image, imageChannel, mapEqual) != 0) {
    fprintf(stderr, "Could not map image channel!\n");
    freeBmpImage(image);
    freeBmpImageChannel(imageChannel);
    return ERROR_EXIT;
  }
  freeBmpImageChannel(imageChannel);

  // Write the image back to disk
  if (saveBmpImage(image, output) != 0) {
    fprintf(stderr, "Could not save output to '%s'!\n", output);
    freeBmpImage(image);
    return ERROR_EXIT;
  };

  ret = 0;
  if (input)
    free(input);
  if (output)
    free(output);
  return ret;
};
